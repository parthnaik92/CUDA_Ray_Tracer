#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <GL/GL.h>
#include <GL/GLU.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <>

#define CRED 0
#define CGREEN 1
#define CBLUE 2

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		getchar();
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/* --------------- VECTORS -------------------- */

struct VECTOR3D{
	double x;
	double y;
	double z;
} ;


/* ----------------- VIEWPORT ----------------- */
struct VIEWPORT {
	int xvmin;
	int yvmin;
	int xvmax;
	int yvmax;
};


/* ------------------- PIXEL ------------------ */
struct PIXEL{
	int i;
	int j;
};


/* ---------------- SPHERE -------------------- */

struct SPHERE_INTERSECTION {
	double	lambda_in;
	double	lambda_out;
	VECTOR3D	normal;
	VECTOR3D point;
	bool	valid;
} ;

struct SPHERE {
	VECTOR3D center;
	double radius;
	double kd_rgb[3];
	double ks_rgb[3];
	double ka_rgb[3];
	double kr_rgb[3];
	double refraction_index;
	double shininess;
	bool mirror;
};


/* ------------------- RAY --------------------- */
struct RAY {
	VECTOR3D origin;
	VECTOR3D direction;
};


/* --------------- VECTOR BASIS ---------------- */
struct VEC_BASIS {
	VECTOR3D u;
	VECTOR3D v;
	VECTOR3D n;
};

__device__ void vec_sub (VECTOR3D *v1, VECTOR3D *v2, VECTOR3D *v3) {

	v1->x = v2->x - v3->x;
	v1->y = v2->y - v3->y;
	v1->z = v2->z - v3->z;
}

__device__ void vec_add (VECTOR3D *v1, VECTOR3D *v2, VECTOR3D *v3) {
	
	v1->x = v2->x + v3->x;
	v1->y = v2->y + v3->y;
	v1->z = v2->z + v3->z;
}

__device__ void vec_scale (double scale, VECTOR3D *v1, VECTOR3D *v2) {
	
	v1->x = scale * v2->x;
	v1->y = scale * v2->y;
	v1->z = scale * v2->z;
}

__device__ double dotproduct (VECTOR3D *v1, VECTOR3D *v2) {
	
	return v1->x * v2->x + v1->y * v2->y + v1->z * v2->z;
}

__device__ VECTOR3D crossProduct(VECTOR3D *v1, VECTOR3D *v2) {
	VECTOR3D temp;
	temp.x =   ( (v1->y * v2->z) - (v1->z * v2->y) );
	temp.y = - ( (v1->x * v2->z) - (v1->z * v2->x) );
	temp.z =   ( (v1->x * v2->y) - (v1->y * v2->x) );
	return temp;
}


__device__ void normalize_vector (VECTOR3D *v) {
	
	double magnitude;
	
	// 1. calculate the magnitude (lerngth):
	magnitude = sqrt( dotproduct(v, v) );
	
	// 2. normalize the vector:
	v->x = v->x / magnitude;
	v->y = v->y / magnitude;
	v->z = v->z / magnitude;
}

__device__ void compute_ray(RAY* ray, VECTOR3D* view_point, VIEWPORT* viewport, PIXEL* pixel, VEC_BASIS* camera_frame, double distance) {
	float u, v;
	VECTOR3D v1, v2, v3, v4, dir;
	
	
	// 1. calculate u and v coordinates of the pixels on the image plane:
	u = (double)(viewport->xvmin) + (double)(pixel->i) + 0.5 ;  
	v = (double)(viewport->yvmin) + (double)(pixel->j) + 0.5 ;  
	
	// 2. calculate ray direction
	
	vec_scale(-distance, &v1, &camera_frame->n);
	vec_scale(u, &v2, &camera_frame->u);
	vec_scale(v, &v3, &camera_frame->v);
	
	ray->origin.x = view_point->x;  
	ray->origin.y = view_point->y;
	ray->origin.z = view_point->z;
	
	vec_add(&v4, &v1, &v2);
	vec_add(&dir, &v4, &v3);
	normalize_vector(&dir);
	
	ray->direction.x = dir.x;
	ray->direction.y = dir.y;
	ray->direction.z = dir.z;
}


__device__ void compute_reflected_ray(RAY* reflected_ray, RAY* incidence_ray, SPHERE_INTERSECTION* intersection) {
	
	double dp1;
	VECTOR3D scaled_normal, reflected_direction;
	
	// calculate dot-product between surface normal and the direction of the incidence ray:
	dp1 = dotproduct(&intersection->normal, &incidence_ray->direction);
	// scale surface normal by 2*dp1:
	dp1 = 2*dp1;
	vec_scale(dp1, &scaled_normal, &intersection->normal);
	
	vec_sub(&reflected_direction, &incidence_ray->direction, &scaled_normal);
	
	reflected_ray->origin=intersection->point;
	reflected_ray->direction=reflected_direction;
}

__device__ void compute_refracted_ray(RAY* refracted_ray, RAY* incidence_ray, SPHERE_INTERSECTION* intersection, SPHERE* intersection_sphere)
{
	VECTOR3D normal_normal = crossProduct(&intersection->normal, &incidence_ray->direction);

	float rotationMatrix[4][4]; 
	float inputMatrix[4]= {incidence_ray->direction.x, incidence_ray->direction.y, incidence_ray->direction.z, 1.0};
	float outputMatrix[4] = {0.0, 0.0, 0.0, 0.0};

	float u = normal_normal.x;
	float v = normal_normal.y;
	float w = normal_normal.z;

	VECTOR3D V1 = incidence_ray->direction;
	VECTOR3D V2 = intersection->normal;
	normalize_vector(&V1);
	normalize_vector(&V2);
	float angle = M_PI/2-acosf(dotproduct(&V1, &V2));
	angle = angle - asinf(intersection_sphere->refraction_index*sinf(angle));

	float L = (u*u + v * v + w * w); 
	float u2 = u * u;     
	float v2 = v * v;     
	float w2 = w * w;       
	rotationMatrix[0][0] = (u2 + (v2 + w2) * cos(angle)) / L;
	rotationMatrix[0][1] = (u * v * (1 - cos(angle)) - w * sqrt(L) * sin(angle)) / L;
	rotationMatrix[0][2] = (u * w * (1 - cos(angle)) + v * sqrt(L) * sin(angle)) / L;
	rotationMatrix[0][3] = 0.0;
	rotationMatrix[1][0] = (u * v * (1 - cos(angle)) + w * sqrt(L) * sin(angle)) / L;
	rotationMatrix[1][1] = (v2 + (u2 + w2) * cos(angle)) / L;
	rotationMatrix[1][2] = (v * w * (1 - cos(angle)) - u * sqrt(L) * sin(angle)) / L;
	rotationMatrix[1][3] = 0.0;
	rotationMatrix[2][0] = (u * w * (1 - cos(angle)) - v * sqrt(L) * sin(angle)) / L;
	rotationMatrix[2][1] = (v * w * (1 - cos(angle)) + u * sqrt(L) * sin(angle)) / L;
	rotationMatrix[2][2] = (w2 + (u2 + v2) * cos(angle)) / L;
	rotationMatrix[2][3] = 0.0;
	rotationMatrix[3][0] = 0.0;
	rotationMatrix[3][1] = 0.0;
	rotationMatrix[3][2] = 0.0;
	rotationMatrix[3][3] = 1.0;

	for(int i = 0; i < 4; i++ )
	{           
		outputMatrix[i] = 0;             
		for(int k = 0; k < 4; k++)
			outputMatrix[i]+= rotationMatrix[i][k] * inputMatrix[k];
	}

	refracted_ray->origin=intersection->point;
	refracted_ray->direction.x=outputMatrix[0];
	refracted_ray->direction.y=outputMatrix[1];
	refracted_ray->direction.z=outputMatrix[2];
}


__device__ void compute_shadow_ray(RAY* ray, SPHERE_INTERSECTION* intersection, VECTOR3D* light) {

	VECTOR3D dir;
	
	// ray origin is in the intersection point
	ray->origin.x = intersection->point.x;
	ray->origin.y = intersection->point.y;
	ray->origin.z = intersection->point.z;
	
	// ray direction is from the intersection point towards the light:
	vec_sub(&dir, light, &intersection->point);
	normalize_vector(&dir);
	
	ray->direction.x = dir.x;
	ray->direction.y = dir.y;
	ray->direction.z = dir.z;
}


__device__ double blinnphong_shading(SPHERE_INTERSECTION *intersection, VECTOR3D* light, VECTOR3D* viewpoint, double kd, double ks, double ka, double p, double intensity, double amb_intensity) {
	
	double color_diffuse = 0.0; 
	double color_specular = 0.0;
	
	VECTOR3D l;
	VECTOR3D h;
	VECTOR3D v;
	
	
	// compute vector v :
	vec_sub(&v, viewpoint, &intersection->point);
	normalize_vector(&v);
	
	// compute vector l :
	vec_sub(&l, light, &intersection->point);
	normalize_vector(&l);
	
	// compute vector h:
	vec_add(&h, &v, &l);
	normalize_vector(&h);
	
	
	// compute the diffuse intensity:
	color_diffuse = kd * intensity * dotproduct(&l, &intersection->normal) ;
	if (color_diffuse < 0.0) color_diffuse = 0.0;
	
	// compute the specular intensity:
	color_specular = ks * intensity * pow (dotproduct(&h, &intersection->normal), p);
	if (color_specular < 0.0) color_specular = 0.0;
	
	return (color_diffuse + color_specular + (ka * amb_intensity));	
}
 

__device__ double shadow(double ka, double amb_intensity) {
	
	return (ka * amb_intensity);	
}


__device__ void set_rgb_array(double* rgb_array, double cred, double cgreen, double cblue) {
	rgb_array[CRED] = cred;
	rgb_array[CGREEN] = cgreen;
	rgb_array[CBLUE] = cblue;
}

__device__ bool sphere_intersection (RAY *ray, SPHERE *sphere, SPHERE_INTERSECTION* intersection) {

	double discriminant;
	double A, B, C;
	double lambda1, lambda2;
	VECTOR3D temp;
	
	A = dotproduct(&ray->direction, &ray->direction);
	
	vec_sub(&temp, &ray->origin, &sphere->center);
	B = 2 * dotproduct(&temp, &ray->direction);
	
	vec_sub(&temp, &ray->origin, &sphere->center);
	C = dotproduct(&temp, &temp) - (sphere->radius * sphere->radius);
	
	discriminant = B*B - 4*A*C;
	
	if (discriminant >= 0) {
		lambda1 = (-B + sqrt(discriminant)) / (2*A);
		lambda2 = (-B - sqrt(discriminant)) / (2*A);
		
		// is the object visible from the eye (lambda1,2>0)
		if (lambda1>=0 && lambda2>=0) {
			if (lambda1 == lambda2) {
				intersection->lambda_in = intersection->lambda_out = lambda1;
			}
			else if (lambda1 < lambda2) {
				intersection->lambda_in  = lambda1;
				intersection->lambda_out = lambda2;
			}
			else {
				intersection->lambda_in  = lambda2;
				intersection->lambda_out = lambda1;
			}
			intersection->valid = true;
			return true;
		}
		else {
			intersection->valid = false;
			return false;
		}
	}
	else {
		intersection->valid = false;
		return false;
	}

}


// Calculate normal vector in the point of intersection:
__device__ void intersection_normal(SPHERE *sphere, SPHERE_INTERSECTION* intersection, RAY* ray) {
	
	double lambda, scale;
	VECTOR3D v1, v2, point, normal;
	
	lambda = intersection->lambda_in;
	
	vec_scale(lambda, &v1, &ray->direction);
	vec_add(&point, &v1, &ray->origin);
	
	intersection->point.x = point.x;
	intersection->point.y = point.y;
	intersection->point.z = point.z;
	
	vec_sub(&v2, &point, &sphere->center);
	
	scale = 1.0 / sphere->radius;
	vec_scale(scale, &normal, &v2);
	
	normalize_vector(&normal);

	intersection->normal.x = normal.x;
	intersection->normal.y = normal.y;
	intersection->normal.z = normal.z;
	
}

__device__ void intersection_exit_normal(SPHERE *sphere, SPHERE_INTERSECTION* intersection, RAY* ray) {
	
	double lambda, scale;
	VECTOR3D v1, v2, point, normal;
	
	lambda = intersection->lambda_out;
	
	vec_scale(lambda, &v1, &ray->direction);
	vec_add(&point, &v1, &ray->origin);
	
	intersection->point.x = point.x;
	intersection->point.y = point.y;
	intersection->point.z = point.z;
	
	vec_sub(&v2, &point, &sphere->center);
	
	scale = 1.0 / sphere->radius;
	vec_scale(scale, &normal, &v2);
	
	normalize_vector(&normal);

	intersection->normal.x = normal.x;
	intersection->normal.y = normal.y;
	intersection->normal.z = normal.z;
	
}


#define NSPHERES 4
#define VIEWPLANE 400
#define WINDOW VIEWPLANE*2
#define FOCALDIST 1000
#define RADIUS 200

GLuint vbo;
void *d_vbo_buffer = NULL;

__device__ VEC_BASIS camera_frame;
__device__ VECTOR3D view_point, static_view_point;
__device__ VECTOR3D light;
__device__ SPHERE sphere[NSPHERES];
__device__ VIEWPORT viewport;

__device__ double focal_distance;
__device__ double color;
__device__ double light_intensity, ambi_light_intensity;

void Timer (int obsolete) {

	glutPostRedisplay();
	glutTimerFunc(30, Timer, 0);
}

void createVBO(GLuint* vbo)
{
	//Create vertex buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	//Initialize VBO
	unsigned int size = (VIEWPLANE<<1) * (VIEWPLANE<<1) * 3 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//Register VBO with CUDA
	cudaGLRegisterBufferObject(*vbo);
}

__device__ float timer=0.0f;

__global__ void animate_kernel()
{
	sphere[0].center.y=static_view_point.y+sinf(timer)*100;
	sphere[1].center.y=static_view_point.y+50+sinf(2*timer)*100;
	sphere[2].center.y=static_view_point.y+100+sinf(1.5*timer)*100;
	//view_point.x=static_view_point.x+sinf(timer)*200;
	timer+=0.02f;
}

union Color
{
	float c;
	uchar4 components;
};

//__device__ __noinline__ void calculateRefraction(double* red, double* green, double* blue, RAY ray, int intersection_object, SPHERE_INTERSECTION current_intersection, double kr, double kg, double kb, int level);

__device__ __noinline__ void calculateReflection(double* red, double* green, double* blue, RAY ray, int intersection_object, SPHERE_INTERSECTION current_intersection, double kr, double kg, double kb, int level)
{
	if (!level)
		return;
	RAY reflected_ray, shadow_ray;
	SPHERE_INTERSECTION reflected_ray_intersection, current_reflected_intersection, shadow_ray_intersection;
	compute_reflected_ray(&reflected_ray, &ray, &current_intersection);
	double reflected_theta = dotproduct(&(reflected_ray.direction), &(current_intersection.normal));
	double current_reflected_lambda = 0x7fefffffffffffff;
	double theta;
	bool bShadow=false;
	int reflected_intersection_object = -1;
	for (int l=0; l<NSPHERES; l++)
	{
		if (l!=intersection_object)
		{
			if (sphere_intersection(&reflected_ray, &sphere[l], &reflected_ray_intersection) && (reflected_theta>0.0))
			{
				intersection_normal(&sphere[l], &reflected_ray_intersection, &reflected_ray);
				if (reflected_ray_intersection.lambda_in<current_reflected_lambda)
				{
					current_reflected_lambda=reflected_ray_intersection.lambda_in;
					reflected_intersection_object=l;
					current_reflected_intersection=reflected_ray_intersection;
				}
			}
		}
	}
	if (reflected_intersection_object>=0)
	{
		compute_shadow_ray(&shadow_ray, &current_reflected_intersection, &light);
		theta = dotproduct(&(shadow_ray.direction), &(current_reflected_intersection.normal));
		for (int l=0; l<NSPHERES; l++)
		{
			if (l!=reflected_intersection_object)
			{
				if (sphere_intersection(&shadow_ray, &sphere[l], &shadow_ray_intersection) && (theta>0.0))
					bShadow=true;
			}
		}
		if (bShadow)
		{
			*red += kr*sphere[intersection_object].ks_rgb[CRED]*shadow(sphere[reflected_intersection_object].ka_rgb[CRED], ambi_light_intensity);
			*green += kg*sphere[intersection_object].ks_rgb[CGREEN]*shadow(sphere[reflected_intersection_object].ka_rgb[CGREEN], ambi_light_intensity);
			*blue += kb*sphere[intersection_object].ks_rgb[CBLUE]*shadow(sphere[reflected_intersection_object].ka_rgb[CBLUE], ambi_light_intensity);
		}
		else
		{
			*red += kr*sphere[intersection_object].ks_rgb[CRED]*blinnphong_shading(&current_reflected_intersection, &light, &view_point, sphere[reflected_intersection_object].kd_rgb[CRED], sphere[reflected_intersection_object].ks_rgb[CRED], sphere[reflected_intersection_object].ka_rgb[CRED], sphere[reflected_intersection_object].shininess,light_intensity, ambi_light_intensity);
			*green += kg*sphere[intersection_object].ks_rgb[CGREEN]*blinnphong_shading(&current_reflected_intersection, &light, &view_point, sphere[reflected_intersection_object].kd_rgb[CGREEN], sphere[reflected_intersection_object].ks_rgb[CGREEN], sphere[reflected_intersection_object].ka_rgb[CGREEN], sphere[reflected_intersection_object].shininess, light_intensity, ambi_light_intensity);
			*blue += kb*sphere[intersection_object].ks_rgb[CBLUE]*blinnphong_shading(&current_reflected_intersection, &light, &view_point,sphere[reflected_intersection_object].kd_rgb[CBLUE], sphere[reflected_intersection_object].ks_rgb[CBLUE], sphere[reflected_intersection_object].ka_rgb[CBLUE], sphere[reflected_intersection_object].shininess, light_intensity, ambi_light_intensity);
		}
		calculateReflection(red, green, blue, reflected_ray, reflected_intersection_object, current_reflected_intersection, kr*sphere[intersection_object].ks_rgb[CRED], kg*sphere[intersection_object].ks_rgb[CGREEN], kb*sphere[intersection_object].ks_rgb[CBLUE], level-1);
		//calculateRefraction(red, green, blue, reflected_ray, reflected_intersection_object, current_reflected_intersection, kr*sphere[intersection_object].kr_rgb[CRED], kg*sphere[intersection_object].kr_rgb[CGREEN], kb*sphere[intersection_object].kr_rgb[CBLUE], level-1);
	}
}

__device__ __noinline__ void calculateRefraction(double* red, double* green, double* blue, RAY ray, int intersection_object, SPHERE_INTERSECTION current_intersection, double kr, double kg, double kb, int level)
{
	if (!level)
		return;
	RAY refracted_ray, shadow_ray;
	SPHERE_INTERSECTION refracted_ray_intersection, shadow_ray_intersection;
	compute_refracted_ray(&refracted_ray, &ray, &current_intersection, &sphere[intersection_object]);
	sphere_intersection(&refracted_ray, &sphere[intersection_object], &refracted_ray_intersection);
	RAY tempRefractedRay = refracted_ray;
	intersection_exit_normal(&sphere[intersection_object], &refracted_ray_intersection, &refracted_ray);
	compute_refracted_ray(&refracted_ray, &tempRefractedRay, &refracted_ray_intersection, &sphere[intersection_object]);
	double current_refracted_lambda = 0x7fefffffffffffff;
	double theta;
	bool bShadow=false;
	int refracted_intersection_object = -1;
	SPHERE_INTERSECTION current_refracted_intersection;
	for (int l=0; l<NSPHERES; l++)
	{
		if (l!=intersection_object)
		{
			if (sphere_intersection(&refracted_ray, &sphere[l], &refracted_ray_intersection))
			{
				intersection_normal(&sphere[l], &refracted_ray_intersection, &refracted_ray);
				if (refracted_ray_intersection.lambda_in<current_refracted_lambda)
				{
					current_refracted_lambda=refracted_ray_intersection.lambda_in;
					refracted_intersection_object=l;
					current_refracted_intersection=refracted_ray_intersection;
				}
			}
		}
	}
	if (refracted_intersection_object>=0)
	{
		compute_shadow_ray(&shadow_ray, &current_refracted_intersection, &light);
		theta = dotproduct(&(shadow_ray.direction), &(current_refracted_intersection.normal));
		for (int l=0; l<NSPHERES; l++)
		{
			if (l!=refracted_intersection_object)
			{
				if (sphere_intersection(&shadow_ray, &sphere[l], &shadow_ray_intersection) && (theta>0.0))
					bShadow=true;
			}
		}
		if (bShadow)
		{
			*red += kr*sphere[intersection_object].kr_rgb[CRED]*shadow(sphere[refracted_intersection_object].ka_rgb[CRED], ambi_light_intensity);
			*green += kg*sphere[intersection_object].kr_rgb[CGREEN]*shadow(sphere[refracted_intersection_object].ka_rgb[CGREEN], ambi_light_intensity);
			*blue += kb*sphere[intersection_object].kr_rgb[CBLUE]*shadow(sphere[refracted_intersection_object].ka_rgb[CBLUE], ambi_light_intensity);
		}
		else
		{
			*red += kr*sphere[intersection_object].kr_rgb[CRED]*blinnphong_shading(&current_refracted_intersection, &light, &view_point, sphere[refracted_intersection_object].kd_rgb[CRED], sphere[refracted_intersection_object].ks_rgb[CRED], sphere[refracted_intersection_object].ka_rgb[CRED], sphere[refracted_intersection_object].shininess,light_intensity, ambi_light_intensity);
			*green += kg*sphere[intersection_object].kr_rgb[CGREEN]*blinnphong_shading(&current_refracted_intersection, &light, &view_point, sphere[refracted_intersection_object].kd_rgb[CGREEN], sphere[refracted_intersection_object].ks_rgb[CGREEN], sphere[refracted_intersection_object].ka_rgb[CGREEN], sphere[refracted_intersection_object].shininess, light_intensity, ambi_light_intensity);
			*blue += kb*sphere[intersection_object].kr_rgb[CBLUE]*blinnphong_shading(&current_refracted_intersection, &light, &view_point,sphere[refracted_intersection_object].kd_rgb[CBLUE], sphere[refracted_intersection_object].ks_rgb[CBLUE], sphere[refracted_intersection_object].ka_rgb[CBLUE], sphere[refracted_intersection_object].shininess, light_intensity, ambi_light_intensity);
		}
		//calculateRefraction(red, green, blue, refracted_ray, refracted_intersection_object, current_refracted_intersection, kr*sphere[intersection_object].kr_rgb[CRED], kg*sphere[intersection_object].kr_rgb[CGREEN], kb*sphere[intersection_object].kr_rgb[CBLUE], level-1);
		//(*calcRefr)(red, green, blue, refracted_ray, refracted_intersection_object, current_refracted_intersection, kr*sphere[intersection_object].kr_rgb[CRED], kg*sphere[intersection_object].kr_rgb[CGREEN], kb*sphere[intersection_object].kr_rgb[CBLUE], level-1);
		calculateReflection(red, green, blue, refracted_ray, refracted_intersection_object, current_refracted_intersection, kr*sphere[intersection_object].ks_rgb[CRED], kg*sphere[intersection_object].ks_rgb[CGREEN], kb*sphere[intersection_object].ks_rgb[CBLUE], level-1);
	}
}

__global__ void init_kernel()
{
	/*calcRefl=calculateReflection;
	calcRefr=calculateRefraction;*/

	// set scene:
	viewport.xvmin = -VIEWPLANE;
	viewport.yvmin = -VIEWPLANE;
	viewport.xvmax = VIEWPLANE;
	viewport.yvmax = VIEWPLANE;
	
	camera_frame.u.x = 1.0;
	camera_frame.u.y = 0.0;
	camera_frame.u.z = 0.0;
	
	camera_frame.v.x = 0.0;
	camera_frame.v.y = 1.0;
	camera_frame.v.z = 0.0;
	
	camera_frame.n.x = 0.0;
	camera_frame.n.y = 0.0;
	camera_frame.n.z = 1.0;
	
	view_point.x = (viewport.xvmax - viewport.xvmin) / 2.0 ;
	view_point.y = (viewport.yvmax - viewport.yvmin) / 2.0 ;
	view_point.z = 0.0;
	static_view_point=view_point;
	
	
	light.x = view_point.x - 1300;
	light.y = view_point.y + 1300;
	light.z = view_point.z - 300;
	
	
	ambi_light_intensity = 1.0;
	light_intensity = 1.0;
	
	focal_distance = FOCALDIST;
	
	
	sphere[0].radius = RADIUS/1.5;
	sphere[0].center.x  = view_point.x - (RADIUS+30);
	sphere[0].center.y  = view_point.y ;
	sphere[0].center.z  = view_point.z - focal_distance - (2*RADIUS+20);
	// the first sphere is blue:
	set_rgb_array(sphere[0].kd_rgb, 0.0, 0.0, 0.8);
	set_rgb_array(sphere[0].ks_rgb, 1.0, 1.0, 1.0);
	set_rgb_array(sphere[0].ka_rgb, 0.0, 0.0, 0.2);
	set_rgb_array(sphere[0].kr_rgb, 0.0, 0.0, 0.0);
	sphere[0].shininess = 100.0;
	sphere[0].refraction_index=1.52;
	sphere[0].mirror = false;
	
	sphere[1].radius = RADIUS/1.2;
	sphere[1].center.x  = view_point.x + 0;
	sphere[1].center.y  = view_point.y + 50;
	sphere[1].center.z  = view_point.z - focal_distance - (3*RADIUS+20);
	// the second sphere is green:
	set_rgb_array(sphere[1].kd_rgb, 0.0, 0.8, 0.0);
	set_rgb_array(sphere[1].ks_rgb, 0.5, 0.5, 0.5);
	set_rgb_array(sphere[1].ka_rgb, 0.0, 0.2, 0.0);
	set_rgb_array(sphere[1].kr_rgb, 0.5, 0.5, 0.5);
	sphere[1].shininess = 10.0;
	sphere[1].refraction_index=1.52;
	sphere[1].mirror = false;
	
	
	sphere[2].radius = RADIUS;
	sphere[2].center.x  = view_point.x + (2*RADIUS+30);
	sphere[2].center.y  = view_point.y + 100;
	sphere[2].center.z  = view_point.z - focal_distance - (4*RADIUS+20);
	// the third sphere is red:
	set_rgb_array(sphere[2].kd_rgb, 0.8, 0.0, 0.0);
	set_rgb_array(sphere[2].ks_rgb, 0.7, 0.7, 0.7);
	set_rgb_array(sphere[2].ka_rgb, 0.2, 0.0, 0.0);
	set_rgb_array(sphere[2].kr_rgb, 0.3, 0.3, 0.3);
	sphere[2].shininess = 100.0;
	sphere[2].refraction_index=1.52;
	sphere[2].mirror = false;
	
	
	sphere[3].radius = 100*RADIUS;
	sphere[3].center.x  = view_point.x ;
	sphere[3].center.y  = view_point.y - 100*RADIUS-130;
	sphere[3].center.z  = view_point.z - focal_distance - (4*RADIUS+20);
	// the third sphere is red:
	set_rgb_array(sphere[3].kd_rgb, 0.2, 0.2, 0.2);
	set_rgb_array(sphere[3].ks_rgb, 0.8, 0.8, 0.5);
	set_rgb_array(sphere[3].ka_rgb, 0.0, 0.0, 0.0);
	set_rgb_array(sphere[3].kr_rgb, 0.2, 0.2, 0.5);
	sphere[3].shininess = 100.0;
	sphere[3].refraction_index=1.52;
	sphere[3].mirror = true;
}

__global__ void rayTrace_kernel(float3* pos)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i>=(viewport.xvmax - viewport.xvmin) || j>(viewport.yvmax - viewport.yvmin))
		return;
	int intersection_object = -1; // none
	int reflected_intersection_object = -1; // none
	double current_lambda = 0x7fefffffffffffff; // maximum positive double
	double current_reflected_lambda = 0x7fefffffffffffff; // maximum positive double

	RAY ray, shadow_ray;
	PIXEL pixel;
	SPHERE_INTERSECTION intersection, current_intersection, shadow_ray_intersection, current_reflected_intersection;

	double red, green, blue;
	double theta, reflected_theta;

	bool bShadow = false;

	pixel.i = i;
	pixel.j = j;
			
	// 1. compute ray:
	compute_ray(&ray, &view_point, &viewport, &pixel, &camera_frame, focal_distance);
			
	// 2. check if ray hits an object:
	for (int k=0; k<NSPHERES; k++)
	{
		if (sphere_intersection(&ray, &sphere[k], &intersection))
		{
			intersection_normal(&sphere[k], &intersection, &ray);
			if (intersection.lambda_in<current_lambda)
			{
				current_lambda=intersection.lambda_in;
				intersection_object=k;
				//copy_intersection_struct(&current_intersection, &intersection);
				current_intersection=intersection;
			}
		}
	}
			
	// Compute the color of the pixel:
	if (intersection_object > -1)
	{
		compute_shadow_ray(&shadow_ray, &current_intersection, &light);
		theta = dotproduct(&(shadow_ray.direction), &(current_intersection.normal));
		for (int l=0; l<NSPHERES; l++)
		{
			if (l!=intersection_object)
			{
				if (sphere_intersection(&shadow_ray, &sphere[l], &shadow_ray_intersection) && (theta>0.0))
					bShadow=true;
			}
		}
		red=green=blue=0;

		// Reflection:
		calculateReflection(&red, &green, &blue, ray, intersection_object, current_intersection, 1.0, 1.0, 1.0, 10);

		// Refraction:
		calculateRefraction(&red, &green, &blue, ray, intersection_object, current_intersection, 1.0, 1.0, 1.0, 10);

		if (bShadow)
		{
			red += shadow(sphere[intersection_object].ka_rgb[CRED], ambi_light_intensity);
			green += shadow(sphere[intersection_object].ka_rgb[CGREEN], ambi_light_intensity);
			blue += shadow(sphere[intersection_object].ka_rgb[CBLUE], ambi_light_intensity);
		}
		else
		{
			red += blinnphong_shading(&current_intersection, &light, &view_point, sphere[intersection_object].kd_rgb[CRED], sphere[intersection_object].ks_rgb[CRED], sphere[intersection_object].ka_rgb[CRED], sphere[intersection_object].shininess, light_intensity, ambi_light_intensity);
			green += blinnphong_shading(&current_intersection, &light, &view_point, sphere[intersection_object].kd_rgb[CGREEN], sphere[intersection_object].ks_rgb[CGREEN], sphere[intersection_object].ka_rgb[CGREEN], sphere[intersection_object].shininess, light_intensity, ambi_light_intensity);
			blue += blinnphong_shading(&current_intersection, &light, &view_point, sphere[intersection_object].kd_rgb[CBLUE], sphere[intersection_object].ks_rgb[CBLUE], sphere[intersection_object].ka_rgb[CBLUE], sphere[intersection_object].shininess, light_intensity, ambi_light_intensity);
		}
		Color temp;
		if (red>1.0)
			red=1.0;
		if (green>1.0)
			green=1.0;
		if (blue>1.0)
			blue=1.0;
		temp.components = make_uchar4((unsigned char)(red*255),(unsigned char)(green*255),(unsigned char)(blue*255),1);
		pos[i*WINDOW+j] = make_float3(i, j, temp.c);
		intersection_object = -1;
		bShadow = false;
	}
	else
	{
		Color temp;
		temp.components = make_uchar4(0,0,0,1);
		pos[i*WINDOW+j] = make_float3(i, j, temp.c);
		intersection_object = -1;
		bShadow = false;
	}
	current_lambda = 0x7fefffffffffffff;
	current_reflected_lambda = 0x7fefffffffffffff;
}

Color* mat, *mat2;
#define ANTI_ALIAS_SIZE 2

__global__ void antiAlias_kernel(/*float3* pos,*/ Color* mat, Color* mat2)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i>ANTI_ALIAS_SIZE*WINDOW || j>ANTI_ALIAS_SIZE*WINDOW)
		return;
	float Kernel[3][3] = {
		{1/9.0, 1/9.0, 1/9.0},
		{1/9.0, 1/9.0, 1/9.0},
		{1/9.0, 1/9.0, 1/9.0}
	};
	double sumX = 0.0, sumY = 0.0, sumZ = 0.0;
	for(int k = -1; k <= 1;++k)
	{
		for(int r = -1; r <=1; ++r)
		{
			sumX += Kernel[r+1][k+1]*mat[(i - r)*WINDOW*ANTI_ALIAS_SIZE+ (j - k)].components.x;
			sumY += Kernel[r+1][k+1]*mat[(i - r)*WINDOW*ANTI_ALIAS_SIZE+ (j - k)].components.y;
			sumZ += Kernel[r+1][k+1]*mat[(i - r)*WINDOW*ANTI_ALIAS_SIZE+ (j - k)].components.z;
		}                 
	}
	Color temp;
	temp.components.x=sumX;
	temp.components.y=sumY;
	temp.components.z=sumZ;
	temp.components.w=mat[i*WINDOW*ANTI_ALIAS_SIZE+j].components.w;
	mat2[i*WINDOW*ANTI_ALIAS_SIZE+j].c=temp.c;//mat[i*WINDOW*ANTI_ALIAS_SIZE+j].c;
}

__global__ void inflate_kernel(float3* pos, Color* mat)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i>WINDOW || j>WINDOW)
		return;
	for (int x=0;x<ANTI_ALIAS_SIZE;++x)
		for (int y=0;y<ANTI_ALIAS_SIZE;++y)
			mat[(ANTI_ALIAS_SIZE*i*WINDOW+x)+ANTI_ALIAS_SIZE*j+y].c=pos[i*WINDOW+j].z;
}

__global__ void deflate_kernel(float3* pos, Color* mat)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i>WINDOW || j>WINDOW)
		return;
	pos[i*WINDOW+j].z=mat[ANTI_ALIAS_SIZE*i*WINDOW+ANTI_ALIAS_SIZE*j].c;
}

void init()
{
	init_kernel<<<1,1>>>();
	hipMalloc(&mat, (ANTI_ALIAS_SIZE*WINDOW)*(ANTI_ALIAS_SIZE*WINDOW) * sizeof(Color));
	hipMalloc(&mat2, (ANTI_ALIAS_SIZE*WINDOW)*(ANTI_ALIAS_SIZE*WINDOW) * sizeof(Color));
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glMatrixMode(GL_PROJECTION);
	gluOrtho2D(0.0, WINDOW, 0.0, WINDOW);
}

void disp(void)
{
	animate_kernel<<<1,1>>>();
	hipDeviceSynchronize();

	float3 *dptr;
    cudaGLMapBufferObject((void**)&dptr, vbo);

	//clear all pixels:
	glClear(GL_COLOR_BUFFER_BIT);
	
	// RAY TRACING:
	dim3 block(32, 16, 1);
	dim3 grid(WINDOW/ block.x, WINDOW / block.y, 1);
	rayTrace_kernel<<<grid,block>>>(dptr);
	HANDLE_ERROR(hipGetLastError());
	hipDeviceSynchronize();
	inflate_kernel<<<grid,block>>>(dptr, mat);
	hipDeviceSynchronize();
	dim3 grid2(ANTI_ALIAS_SIZE*WINDOW/ block.x, ANTI_ALIAS_SIZE*WINDOW / block.y, 1);
	antiAlias_kernel<<<grid2,block>>>(/*dptr, */mat, mat2);
	hipDeviceSynchronize();
	deflate_kernel<<<grid,block>>>(dptr, mat2);
	hipDeviceSynchronize();
	cudaGLUnmapBufferObject(vbo);
	//glFlush();
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4,GL_UNSIGNED_BYTE,12,(GLvoid*)8);

    glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glDrawArrays(GL_POINTS, 0, WINDOW * WINDOW);
    glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();
	//glutPostRedisplay();
}


int main (int argc, char** argv)
{
	// init glut:
	glutInit (&argc, argv);
	// specify the display mode to be RGB and single buffering:
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	// specify the initial window position:
	glutInitWindowPosition(100,100);
	// specify the initial window size:
	glutInitWindowSize(WINDOW,WINDOW);
	// create the window and set title:
	glutCreateWindow("Basic Ray Tracer");
	// init opengl:
	init();
	// register callback function to display graphics:
	glutDisplayFunc(disp);
	glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 ")) {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
		exit(0);
    }
	// call Timer():
	Timer(0);
	createVBO(&vbo);
	// enter tha main loop and process events:
	glutMainLoop();
	hipFree(mat);
	hipFree(mat2);
	return 0;
}